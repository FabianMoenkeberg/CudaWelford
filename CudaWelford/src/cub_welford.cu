#include "test_util.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>

#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>

#include <iostream>
#include <vector>

using namespace hipcub;

struct CustomMin
    {
        template <typename T>
        __device__ __forceinline__
        T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};

struct Pow2
{
    __host__ __device__ __forceinline__
    float operator()(const float &a) const {
        return float(a * a);
    }
};

struct SubstractPow
{
    float constant = 0.0f;

    __host__ __device__ __forceinline__
    SubstractPow(float constant) : constant(constant) {}

    __host__ __device__ __forceinline__
    float operator()(const float &a) const {
        return (a-constant)*(a-constant);
    }
};

void cubBaseAlgorithm(const std::vector<float>& input, float& mean, float& var){
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    int N = input.size();

    // Allocate device memory for input
    float* d_input;
    hipMalloc((void**)&d_input, sizeof(float) * N);

    // Copy input data from host to device
    hipMemcpy(d_input, input.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    // Compute the mean
    
    float* dmean ;
    hipMalloc((void**)&dmean, sizeof(float) );
    // Copy input data from host to device

    // output on host side
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;

    float* dout ;
    hipMalloc((void**)&dout, sizeof(float)*1 );

    gpu_timer.Start();
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_input, dmean, N));

    hipDeviceSynchronize();

    // CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    hipMalloc((void**)&d_temp_storage, temp_storage_bytes);

    hipDeviceSynchronize();

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_input, dmean, N);

    hipDeviceSynchronize();
    hipMemcpy(&mean, dmean, sizeof(float) * 1, hipMemcpyDeviceToHost);

    mean /= N;
    SubstractPow subPow_op(mean);

    CacheModifiedInputIterator<LOAD_LDG,float> cached_iter(d_input);
    TransformInputIterator<float, SubstractPow, CacheModifiedInputIterator<LOAD_LDG, float> > input_iter(cached_iter, subPow_op);
    // hipcub::TransformInputIterator<float, SubstractPow, float*> input_iter(d_input, subPow_op);

    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, input_iter, dout, N, Sum(), 0);
    
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    // Run reduction
    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, input_iter, dout, N, Sum(), 0);

    hipDeviceSynchronize();
    gpu_timer.Stop();
    elapsed_millis = gpu_timer.ElapsedMillis();

    hipDeviceSynchronize();
    printf("Run time: %f\n", elapsed_millis);

    var = 0;
    hipMemcpy(&var, dout, sizeof(float) * 1, hipMemcpyDeviceToHost);

    var/=(N-1);

    // Cleanup
    hipFree(d_input);
    hipFree(d_temp_storage);
    hipFree(dout);
    hipFree(dmean);
}

int cubWelford() {
    
    const int N = 1024*1024*2;
    std::vector<float> input(N);// = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

    float sum = 0;
    for (int i = 0; i < N ; i++) {
        input[i] = 1.0f + 100*(float)rand()/(float)RAND_MAX;
        input[i] = static_cast<float>(i%2);
        sum+= input[i];
    }

    float mean = 0;
    float var = 0;

    cubBaseAlgorithm(input, mean, var);
    

    printf("Mean: %f, Var: %f\n", mean, var);
    std::cout << "Variance: " << mean << std::endl;

    return 0;
}
