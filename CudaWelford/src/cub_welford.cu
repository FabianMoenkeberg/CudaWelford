#include "../include/cub_welford.h"
#include "../include/cub_sum.h"

#include "../include/test_util.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>

#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>

#include "cub_sum.h"

#include <iostream>
#include <vector>

using namespace hipcub;
using namespace CubSum;

struct CustomMin
    {
        template <typename T>
        __device__ __forceinline__
        T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};

struct Pow2
{
    __host__ __device__ __forceinline__
    float operator()(const float &a) const {
        return float(a * a);
    }
};

/// @brief Fast way to calculate (a-const)^2 on a single cub transformation call.
/// constant is set in the constructor.
struct SubstractPow
{
    float constant = 0.0f;

    __host__ __device__ __forceinline__
    SubstractPow(float constant) : constant(constant) {}

    __host__ __device__ __forceinline__
    float operator()(const float &a) const {
        return (a-constant)*(a-constant);
    }
};

/// @brief Classical Multicall Method to calculate the Variance with CUB. 
/// @param input vector from which we calculate variance.
/// @param mean     Resulting mean
/// @param var      Resulting variance
void cubBaseAlgorithm(const std::vector<float>& input, float& mean, float& var){
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    int N = input.size();

    // Allocate device memory for input
    float* d_input;
    hipMalloc((void**)&d_input, sizeof(float) * N);

    // Copy input data from host to device
    hipMemcpy(d_input, input.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    // Initialize mean on device
    
    float* dmean ;
    hipMalloc((void**)&dmean, sizeof(float) );
    // Copy input data from host to device

    // output on host side
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;

    float* dout ;
    hipMalloc((void**)&dout, sizeof(float)*1 );

    gpu_timer.Start();
    // Determine temporary storage size with nullptr
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_input, dmean, N));

    hipDeviceSynchronize();

    hipMalloc((void**)&d_temp_storage, temp_storage_bytes);

    hipDeviceSynchronize();

    // 1. Run Reduction to Calculate Sum to calculate Mean value after.
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_input, dmean, N);

    hipDeviceSynchronize();
    // Copy sum from device to host
    hipMemcpy(&mean, dmean, sizeof(float) * 1, hipMemcpyDeviceToHost);

    mean /= N;
    // Initialize Subtract and power operator
    SubstractPow subPow_op(mean);

    // 2. Apply Subtraction of mean to each value and take the value to the power of 2.
    CacheModifiedInputIterator<LOAD_LDG,float> cached_iter(d_input);
    TransformInputIterator<float, SubstractPow, CacheModifiedInputIterator<LOAD_LDG, float> > input_iter(cached_iter, subPow_op);
    // hipcub::TransformInputIterator<float, SubstractPow, float*> input_iter(d_input, subPow_op);

    // Determine temporary storage size with nullptr
    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, input_iter, dout, N, Sum(), 0);
    
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    // 3. Run Reduction to Calculate Sum of the vector and afterwards mean.
    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, input_iter, dout, N, Sum(), 0);

    hipDeviceSynchronize();
    gpu_timer.Stop();
    elapsed_millis = gpu_timer.ElapsedMillis();

    hipDeviceSynchronize();
    printf("Run time: %f\n", elapsed_millis);

    var = 0;
    // Copy result from device to host
    hipMemcpy(&var, dout, sizeof(float) * 1, hipMemcpyDeviceToHost);

    var/=(N-1);

    // Cleanup
    hipFree(d_input);
    hipFree(d_temp_storage);
    hipFree(dout);
    hipFree(dmean);
}

/// @brief Reduction operator to calculate the variance in a single pass using the Welford algorithm.
/// Similar to CustomSum operator in cub_sum.cu
struct WelfordOp
{
    __device__ __forceinline__
    point operator()(const point &a, const point &b) const {
        if (a.N == 0){
            return b;
        }
        float diff = (a.N/b.N*b.T-a.T);
        point res{b.M + a.M + b.N*diff*diff/((b.N+a.N)*a.N), b.T + a.T, a.N + b.N};
        return res;
    }
};


void cubReduceAlgorithm(const std::vector<float>& input0, float& sumOut, float& Nout, float& varNOut){
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    int N = input0.size();
    std::vector<point> input(N);
    for (int i = 0; i < N;++i){
        input.at(i) = point{0, input0.at(i), 1};
    }
    printf("N is %d.\n", N);
    // Allocate device memory for input
    point* d_input;
    hipMalloc((void**)&d_input, sizeof(point) * N);

    // Copy input data from host to device
    hipMemcpy(d_input, input.data(), sizeof(point) * N,hipMemcpyHostToDevice);

    // Compute the mean
    WelfordOp wel_op;
    point init{0, 0, 0}; 
    point sum;
    point dinit;
    
    point* dsum ;
    hipMalloc((void**)&dsum, sizeof(point) );
    hipMalloc((void**)&dinit, sizeof(point) );
    // Copy input data from host to device
    hipMemcpy(&dinit, &init, sizeof(point) * 1, hipMemcpyHostToDevice);

    // output on host side
    void            *d_temp_storage = nullptr;
    size_t          temp_storage_bytes = 0;

    float* dout ;
    hipMalloc((void**)&dout, sizeof(float)*1 );

    gpu_timer.Start();
    // Determine temporary storage size with nullptr and allocate it
    CubDebugExit(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_input, dsum, N, wel_op, init));

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run Welford Reduction
    CubDebugExit(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_input, dsum, N, wel_op, init));

    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(&sum, dsum, sizeof(point) * 1, hipMemcpyDeviceToHost);
    printf(" %f, %f, %f. \n", sum.M, sum.T,sum.N);
    sumOut = sum.T;
    Nout = sum.N;
    varNOut = sum.M;

    // Measure time
    hipDeviceSynchronize();
    gpu_timer.Stop();
    elapsed_millis = gpu_timer.ElapsedMillis();

    hipDeviceSynchronize();
    printf("Run time: %f\n", elapsed_millis);

    // Cleanup
    hipFree(d_input);
    hipFree(d_temp_storage);
    hipFree(dout);
    hipFree(dsum);
}

int cubVarianceReduceMultiCall() {
    
    const int N = 1024*1024*2;
    // const int N = 8;
    std::vector<float> input(N);// = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

    float sum = 0;
    for (int i = 0; i < N ; i++) {
        input[i] = 1.0f + 100*(float)rand()/(float)RAND_MAX;
        input[i] = static_cast<float>(i%2);
        sum+= input[i];
    }

    float mean = 0;
    float var = 0;
    float nOut = 0;

    cubBaseAlgorithm(input, mean, var);

    printf("Mean: %f, Var: %f\n", mean, var);
    std::cout << "Variance: " << mean << std::endl;

    return 0;
}

int cubWelfordReduceSingle() {
    
    const int N = 1024*1024*2;
    // const int N = 8;
    std::vector<float> input(N);// = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

    float sum = 0;
    for (int i = 0; i < N ; i++) {
        input[i] = 1.0f + 100*(float)rand()/(float)RAND_MAX;
        input[i] = static_cast<float>(i%2);
        sum+= input[i];
    }

    float mean = 0;
    float var = 0;
    float nOut = 0;

    cubReduceAlgorithm(input, mean, nOut, var);


    printf("Mean: %f, Var: %f, pure: %f, %f, %d\n", mean/N, var/N, mean, var, N);
    std::cout << "Variance: " << mean << std::endl;

    return 0;
}
