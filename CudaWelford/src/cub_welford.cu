#include "../include/cub_welford.h"
#include "../include/cub_sum.h"

#include "../include/test_util.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>

#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>

#include "cub_sum.h"

#include <iostream>
#include <vector>

using namespace hipcub;

struct CustomMin
    {
        template <typename T>
        __device__ __forceinline__
        T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};

struct Pow2
{
    __host__ __device__ __forceinline__
    float operator()(const float &a) const {
        return float(a * a);
    }
};

struct SubstractPow
{
    float constant = 0.0f;

    __host__ __device__ __forceinline__
    SubstractPow(float constant) : constant(constant) {}

    __host__ __device__ __forceinline__
    float operator()(const float &a) const {
        return (a-constant)*(a-constant);
    }
};

void cubBaseAlgorithm(const std::vector<float>& input, float& mean, float& var){
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    int N = input.size();

    // Allocate device memory for input
    float* d_input;
    hipMalloc((void**)&d_input, sizeof(float) * N);

    // Copy input data from host to device
    hipMemcpy(d_input, input.data(), sizeof(float) * N, hipMemcpyHostToDevice);

    // Compute the mean
    
    float* dmean ;
    hipMalloc((void**)&dmean, sizeof(float) );
    // Copy input data from host to device

    // output on host side
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;

    float* dout ;
    hipMalloc((void**)&dout, sizeof(float)*1 );

    gpu_timer.Start();
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_input, dmean, N));

    hipDeviceSynchronize();

    // CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));
    hipMalloc((void**)&d_temp_storage, temp_storage_bytes);

    hipDeviceSynchronize();

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_input, dmean, N);

    hipDeviceSynchronize();
    hipMemcpy(&mean, dmean, sizeof(float) * 1, hipMemcpyDeviceToHost);

    mean /= N;
    SubstractPow subPow_op(mean);

    CacheModifiedInputIterator<LOAD_LDG,float> cached_iter(d_input);
    TransformInputIterator<float, SubstractPow, CacheModifiedInputIterator<LOAD_LDG, float> > input_iter(cached_iter, subPow_op);
    // hipcub::TransformInputIterator<float, SubstractPow, float*> input_iter(d_input, subPow_op);

    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, input_iter, dout, N, Sum(), 0);
    
    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    
    // Run reduction
    hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, input_iter, dout, N, Sum(), 0);

    hipDeviceSynchronize();
    gpu_timer.Stop();
    elapsed_millis = gpu_timer.ElapsedMillis();

    hipDeviceSynchronize();
    printf("Run time: %f\n", elapsed_millis);

    var = 0;
    hipMemcpy(&var, dout, sizeof(float) * 1, hipMemcpyDeviceToHost);

    var/=(N-1);

    // Cleanup
    hipFree(d_input);
    hipFree(d_temp_storage);
    hipFree(dout);
    hipFree(dmean);
}

struct WelfordOp
{
    __device__ __forceinline__
    point operator()(const point &a, const point &b) const {
        if (a.N == 0){
            return b;
        }
        float diff = (a.N/b.N*b.T-a.T);
        // printf(" %f, %f, %f. \n", a.M, a.T, a.N);
        // printf(" %f, %f, %f. => %f \n", b.M, b.T, b.N, diff);
        // point res{a.M + b.M + a.N*diff*diff/((a.N+b.N)*b.N), a.T + b.T, a.N + b.N};
        point res{b.M + a.M + b.N*diff*diff/((b.N+a.N)*a.N), b.T + a.T, a.N + b.N};
        // printf(" %f, %f, %f. end\n", res.M, res.T, res.N);
        return res;
    }
};

void cubReduceAlgorithm(const std::vector<float>& input0, float& sumOut, float& Nout, float& varNOut){
    GpuTimer gpu_timer;
    float elapsed_millis = 0.0;
    int N = input0.size();
    std::vector<point> input(N);
    for (int i = 0; i < N;++i){
        input.at(i) = point{0, input0.at(i), 1};
    }
    printf("N is %d.\n", N);
    // Allocate device memory for input
    point* d_input;
    hipMalloc((void**)&d_input, sizeof(point) * N);

    // Copy input data from host to device
    hipMemcpy(d_input, input.data(), sizeof(point) * N,hipMemcpyHostToDevice);

    // Compute the mean
    WelfordOp wel_op;
    point init{0, 0, 0}; 
    point sum;
    point dinit;
    
    point* dsum ;
    hipMalloc((void**)&dsum, sizeof(point) );
    hipMalloc((void**)&dinit, sizeof(point) );
    // Copy input data from host to device
    hipMemcpy(&dinit, &init, sizeof(point) * 1, hipMemcpyHostToDevice);

    // output on host side
    void            *d_temp_storage = nullptr;
    size_t          temp_storage_bytes = 0;

    float* dout ;
    hipMalloc((void**)&dout, sizeof(float)*1 );

    gpu_timer.Start();
    // Determine temporary storage size with nullptr
    CubDebugExit(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_input, dsum, N, wel_op, init));

    hipMalloc(&d_temp_storage, temp_storage_bytes);
    // Run Reduction
    CubDebugExit(hipcub::DeviceReduce::Reduce(d_temp_storage, temp_storage_bytes, d_input, dsum, N, wel_op, init));

    hipDeviceSynchronize();
    hipMemcpy(&sum, dsum, sizeof(point) * 1, hipMemcpyDeviceToHost);
    printf(" %f, %f, %f. \n", sum.M, sum.T,sum.N);
    sumOut = sum.T;
    Nout = sum.N;
    varNOut = sum.M;

    hipDeviceSynchronize();
    gpu_timer.Stop();
    elapsed_millis = gpu_timer.ElapsedMillis();

    hipDeviceSynchronize();
    printf("Run time: %f\n", elapsed_millis);

    // Cleanup
    hipFree(d_input);
    hipFree(d_temp_storage);
    hipFree(dout);
    hipFree(dsum);
}

int cubWelfordReduceMultiCall() {
    
    const int N = 1024*1024*2;
    // const int N = 8;
    std::vector<float> input(N);// = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

    float sum = 0;
    for (int i = 0; i < N ; i++) {
        input[i] = 1.0f + 100*(float)rand()/(float)RAND_MAX;
        input[i] = static_cast<float>(i%2);
        sum+= input[i];
    }

    float mean = 0;
    float var = 0;
    float nOut = 0;

    cubBaseAlgorithm(input, mean, var);

    printf("Mean: %f, Var: %f\n", mean, var);
    std::cout << "Variance: " << mean << std::endl;

    return 0;
}

int cubWelfordReduceSingle() {
    
    const int N = 1024*1024*2;
    // const int N = 8;
    std::vector<float> input(N);// = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};

    float sum = 0;
    for (int i = 0; i < N ; i++) {
        input[i] = 1.0f + 100*(float)rand()/(float)RAND_MAX;
        input[i] = static_cast<float>(i%2);
        sum+= input[i];
    }

    float mean = 0;
    float var = 0;
    float nOut = 0;

    cubReduceAlgorithm(input, mean, nOut, var);


    printf("Mean: %f, Var: %f, pure: %f, %f, %d\n", mean/N, var/N, mean, var, N);
    std::cout << "Variance: " << mean << std::endl;

    return 0;
}
