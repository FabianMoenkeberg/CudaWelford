#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>

#include <stdio.h>
#include <limits.h>
#include <iostream>

bool checkResults(float& mean, float &var, float& mean2, float& var2, float rel_tol) {
    printf("mean = %f, var = %f and mean_ref = %f and var_ref = %f \n", mean, var, mean2, var2);

    if (abs(mean-mean2)/mean > rel_tol|| abs(var-var2)/var > rel_tol) {
    printf("Error solutions don't match \n");
    return false;
    }
  
  return true;
}

void computeCpuStatistics(float *g_data, int dimx, float& mean, float& var, int iStart, int iEnd) {

float T = 0;
float T0 = 0;
float M = 0;
long N = 0;
long N0 = 1;
// #pragma omp parallel for
    for (int ix = iStart; ix < iEnd; ++ix) {
      int idx = ix;

      float value = g_data[idx];
      T0 = T;
      T += value;
      N++;
      M += (value - T/N)*(value - T0/N0);
      N0 = N;
    }
    printf("Result %f, %d \n", M, N);
    mean = T/N;
    var = M/(N-1);
}


void computeCpuStatisticsTest(float *g_data, int dimx, float& mean, float& var) {

float T = 0;
float T0 = 0;
float M = 0;
long N = 0;
// #pragma omp parallel for
    for (int ix = 0; ix < dimx; ++ix) {
      int idx = ix;

      float value = g_data[idx];
      T += value;
      N++;
    }

    mean = T/N;
    float diff;
    for (int ix = 0; ix < dimx; ++ix) {
      int idx = ix;

      float value = g_data[idx];
      // printf("Result %f.\n", value);
      diff = value - mean;
      M += diff*diff;
    }
    
    printf("Result Test %f, %d, %f \n", M, N, mean);
    var = M/(N-1);
}

__inline__ __device__ void warpReduceWelford(float& T0in, float& T2in, float& Min, float& M2in, int warpSize, int n) {
  float T0 = T0in;
  float T2 = T2in;
  float M = Min;
  float M2 = M2in;
  float diff;
  // if (n>1024) printf("Warp00 %d: M2 %f, T2 %f, M %f, T %f\n", threadIdx.x, M2, T2, M, T0);
  for (int offset = warpSize/2; offset > 0; offset /= 2){ 
    diff = (T0 - T2);
    // printf("Diff %d, diff %f, M %f, M2 %f, n %d, %f\n", threadIdx.x, diff, M, M2, n);
    M += M2 + diff*diff/(2*n);
    T0 += T2;
    // printf("Warp0 %d: M2 %f, T2 %f, M %f, T %f\n", threadIdx.x, M2, T2, M, T0);
    // __syncthreads();
    T2 = __shfl_down_sync(0xffffffff, T0, static_cast<unsigned int>(offset), warpSize);
    M2 = __shfl_down_sync(0xffffffff, M, static_cast<unsigned int>(offset), warpSize);
    // printf("Warp %d: M2 %f, T2 %f, M %f, T %f\n", threadIdx.x, M2, T2, M, T0);
    n*=2;
  }

  diff = (T0 - T2);
  M += M2 + diff*diff/(2*n);
  T0 += T2;

  T0in = T0;
  Min = M;
}

// Note: try to run her again with just smaller size and different structure. See output and take it as input
__global__ void kernelWelfordWarp(float *g_data, float *g_out, int n0, bool firstRun) {
 
    extern __shared__ float sdata[];

    int Nhalf = blockDim.x;
    int Nhalf0 = Nhalf;
    unsigned int tid = threadIdx.x;

    int idx =  blockIdx.x * (blockDim.x) + tid;
    int idx2 = tid + Nhalf;
    int dT = Nhalf;
    float T, T2, diffT;
    float M = 0.0f;
    float M2 = 0.0f;
    
    if (!firstRun){
      M = g_out[idx]; // e.g. M
      M2 = g_out[idx2];
    }

    T = g_data[idx]; // e.g. M
    T2 = g_data[idx2];
    
    int lane = tid % warpSize;
    int wid = threadIdx.x / warpSize;
    
    warpReduceWelford(T, T2, M, M2, warpSize, 1);
    
    if (lane==0) {
      sdata[wid] = M;
      sdata[wid + dT] = T;
    }
    n0 = 2*warpSize;
    
    Nhalf/=n0;
    
    __syncthreads();
    if (!firstRun){
      M = sdata[tid];  
      M2 = sdata[tid + Nhalf];
    }else{
      M = sdata[tid];
      M2 = sdata[tid + Nhalf];
    }
    
    while (Nhalf>0){
        idx2 = tid + Nhalf;
        
        if (tid < Nhalf)
        {
          // calculateOneWelfordStep(sdata, T, T2, diffT, M, M2);
          T = sdata[tid+dT];
          T2 = sdata[idx2+dT];

          diffT = (T - T2);
          
          M += M2 + diffT*diffT/(2*n0);
          sdata[tid] = M;
          T += T2;
          
          sdata[tid+dT] = T;
        }
        n0*=2;
        Nhalf/=2;
        __syncthreads();
        M2 = sdata[tid + Nhalf];
    }
    if (tid == 0){
      g_data[blockIdx.x] = T;//(blockDim.x*2);
      // printf("Results kernel T: %f, %d, %f, %d \n", T, blockDim.x*2, g_out[blockIdx.x+gridDim.x], blockIdx.x);
      g_out[blockIdx.x] = M;//(blockDim.x*2 - 1);
      // printf("Results kernel M: %f, %d, %f %d \n", M, gridDim.x, g_out[blockIdx.x], blockIdx.x);
    }
}

// Note: try to run her again with just smaller size and different structure. See output and take it as input
__global__ void kernelWelford2B(float *g_data, float *g_out, int n0, bool firstRun) {
    extern __shared__ float sdata[];
    int diff = n0;
    int N = 2*blockDim.x;
    
    int Nhalf = N/2;
    int Nhalf0 = Nhalf;
    unsigned int tid = threadIdx.x;

    int idx =  blockIdx.x * (blockDim.x) + tid;
    int idx2 = tid + Nhalf;
    int dT = Nhalf;
    float M, M2, T, T2, T0;
    M = 0;
    M2 = 0;
    if (firstRun){
      sdata[tid] = g_data[idx2]; // e.g. M
      dT = 0;
    }else{
      sdata[tid] = g_out[idx]; // e.g. M
    }
    
    T = g_data[idx]; // e.g. T
    sdata[tid + Nhalf] = T;
    // printf("Idx %d, %d, %d, %d\n", idx, dimx, dT, Nhalf);
    // printf("Init %f, %f\n", sdata[tid], sdata[tid + Nhalf]);
    __syncthreads();
    if (!firstRun){
      M = sdata[tid];
      N/=2;
      Nhalf/=2;
      M2 = sdata[tid + Nhalf];
    }
    
    
    while (Nhalf>0){
        idx2 = tid + Nhalf;
        
        if (tid < Nhalf)
        {
          T = sdata[tid+dT];
          T2 = sdata[idx2+dT];
          // printf("Values %f, %f, %f, %f\n", M, M2, T, T2);
          T0 = (T - T2);
          
          M = M + M2 + T0*T0/(2*diff);
          sdata[tid] = M;
          T += T2;
          dT = Nhalf0;
          sdata[tid+dT] = T;
        }
        diff*=2;
        N /= 2;
        Nhalf = N/2;
        __syncthreads();
        M2 = sdata[tid + Nhalf];
    }
    if (tid == 0){
      g_data[blockIdx.x] = T;//(blockDim.x*2);
      // printf("Results kernel T: %f, %d, %f, %d \n", T, blockDim.x*2, g_out[blockIdx.x+gridDim.x], blockIdx.x);
      g_out[blockIdx.x] = M;//(blockDim.x*2 - 1);
      // printf("Results kernel M: %f, %d, %f %d \n", M, gridDim.x, g_out[blockIdx.x], blockIdx.x);
    }
}

__global__ void kernelWelford(float *g_data, float *g_out, int dimx) {
    int diff = 1;
    int N = 2*blockDim.x;
    
    int Nhalf = N/2;
    int Nhalf0 = Nhalf;
    int x = blockIdx.x * (blockDim.x*2) + threadIdx.x;
    int idx = (x);
    int idx2 = idx + Nhalf;
    int dT = 0;
    float M, M2, T, T2, T0;
    M = 0;
    M2 = 0;
    
    // printf("Block: %d, %d, %d \n", blockIdx.x, idx, idx2);
    while (N>0){
        idx2 = idx + Nhalf;
        
        if (threadIdx.x < Nhalf)
        {
          T = g_data[idx+dT];
          T2 = g_data[idx2+dT];
          T0 = (T - T2);

          // if (blockIdx.x==1){
          //   printf("IDX %f, %f, %d, %d: %f, %f, %f: %f, %f : %f \n", g_data[idx+dT], g_data[idx2+dT], idx+dT, idx2+dT, T, T2, T0, M, M2, M + M2 + T0*T0/(2*diff));
          // }           
          
          M = M + M2 + T0*T0/(2*diff);
          g_data[idx] = M;
          T += T2;
          dT = Nhalf0;
          g_data[idx+dT] = T;
        }
        diff*=2;
        N /= 2;
        Nhalf = N/2;
        __syncthreads();
        M2 = g_data[idx+Nhalf];
    }
    if (threadIdx.x == 0){
      g_out[blockIdx.x+gridDim.x] = T;//(blockDim.x*2);
      // printf("Results kernel T: %f, %d, %f, %d \n", T, blockDim.x*2, g_out[blockIdx.x+dimx/2], blockIdx.x);
      g_out[blockIdx.x] = M;//(blockDim.x*2 - 1);
      // printf("Results kernel M: %f, %d, %f %d \n", M, dimx, g_out[blockIdx.x], blockIdx.x);
    }
}


void launchKernelWelford(float * d_data, float *d_out, int dimx, int& nBlocks) {
  // Only change the contents of this function and the kernel(s). You may
  // change the kernel's function signature as you see fit. 

  //query number of SMs
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  int num_sms = prop.multiProcessorCount;
  int blockSize = min(1024, dimx/2);
  nBlocks = dimx/2/blockSize;
  dim3 block(blockSize, 1);
  dim3 grid(nBlocks, 1);
  // printf("test00 %d", blockSize);
  kernelWelfordWarp<<<grid, block, blockSize*2*sizeof(float)>>>(d_data, d_out, 1, true);
  // printf("test000 %d", nBlocks); 
  dimx = nBlocks;
  blockSize = min(1024, dimx/2);
  // printf("Blocksize: %d\n", blockSize);
  nBlocks = dimx/blockSize/2;
  block.x = blockSize;
  grid.x = nBlocks;

  kernelWelfordWarp<<<grid, block, blockSize*2*sizeof(float)>>>(d_data, d_out, 2*1024, false);
  // kernelWelford2B<<<grid, block, blockSize*2*sizeof(float)>>>(d_data, d_out, 1024, false);


  // kernelWelford<<<grid, block>>>(d_data, d_out, dimx);
}



float algorithmWelford(float *d_data, float *d_out, int dimx, int& nBlocks) {
  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  // int nBlocks = 0;
  launchKernelWelford(d_data, d_out, dimx, nBlocks);
  
  // printf("number of blocks: %d \n", nBlocks);
  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return elapsed_time_ms;
}

void calcRemainingVar(float* h_data, float* h_out, int dimx, int nBlocks, float* totvar, float* totmean){
  *totvar = h_out[0];
  *totmean = h_data[0];
  float diff;
  int n = dimx/nBlocks; 
  float m = static_cast<float>(dimx/nBlocks);

  for(int i = 0; i < nBlocks; ++i){
    printf("Results kernel %d: M %f,  T %f, N %d \n", i, h_out[i], h_data[i], dimx/nBlocks);
  }
  for(int i = 1; i < nBlocks; ++i){
    n+= dimx/nBlocks;
    diff = (*totmean - (n-m)*h_data[i]/m);
    // printf("diff: %f, %f, %f, %f\n", diff, h_data[i], totvar, h_data[i] +m*diff*diff/(n-m)/n);
    *totvar += h_out[i] +m*diff*diff/(n-m)/n;
    *totmean+=h_data[i];
  }
}

int run_welford() {
  int dimx = 1024*1024*2;

  int nbytes = dimx * sizeof(float);

  float *d_data = 0, *h_data = 0, *h_out, *h_gold = 0, *d_out = 0;
  hipMalloc((void **)&d_data, nbytes);
  hipMalloc((void **)&d_out, nbytes);
  if (0 == d_data) {
    printf("couldn't allocate GPU memory\n");
    return -1;
  }
  printf("allocated %.2f MB on GPU\n", nbytes / (1024.f * 1024.f));
  h_data = (float *)malloc(nbytes);
  h_out = (float *)malloc(nbytes);
  h_gold = (float *)malloc(nbytes);
  if (0 == h_data || 0 == h_gold) {
    printf("couldn't allocate CPU memory\n");
    return -2;
  }
  printf("allocated %.2f MB on CPU\n", 2.0f * nbytes / (1024.f * 1024.f));
  float sum = 0;
  for (int i = 0; i < dimx ; i++) {
    h_gold[i] = 1.0f + 100*(float)rand()/(float)RAND_MAX;
    h_gold[i] = 1.0f + static_cast<float>(i%2);
    sum+= h_gold[i];
  }
  printf("sum vector: %f\n", sum);

  hipMemcpy(d_data, h_gold, nbytes, hipMemcpyHostToDevice);
  
  int nBlocks = 0;
  float time = algorithmWelford(d_data, d_out, dimx, nBlocks);

  hipMemcpy(h_data, d_data, nbytes, hipMemcpyDeviceToHost);
  hipMemcpy(h_out, d_out, nbytes, hipMemcpyDeviceToHost);
  float var0 = 0;
  float mean0 = 0;
  float* totvar = &var0;
  float* totmean = &mean0;
  calcRemainingVar(h_data, h_out, dimx, nBlocks, totvar, totmean);

  float meanCuda = *totmean/dimx;
  float varCuda = *totvar/(dimx-1);
  printf("Runtime Cuda: %f\n", time);
  printf("Mean %f, and Var %f \n", meanCuda, varCuda);
  printf("Verifying solution\n");

//   hipMemcpy(h_data, d_data, nbytes, hipMemcpyDeviceToHost);

  float rel_tol = .001;
  float mean = 0;
  float var = 0;
  computeCpuStatistics(h_gold, dimx, mean, var, 0, dimx);
  float mean2 = 0;
  float var2 = 0;
  computeCpuStatisticsTest(h_gold, dimx, mean2, var2);
  bool passCuda = checkResults(meanCuda, varCuda, mean2, var2, rel_tol);
  bool pass = checkResults(mean, var, mean2, var2, rel_tol);

  if (pass && passCuda) {
    printf("Results are correct\n");
  } else {
    printf("FAIL:  results are incorrect\n");
  }  

  float elapsed_time_ms = 0.0f;
 
//   elapsed_time_ms = timing_experiment(d_data, dimx, dimy, niterations, nreps);
  printf("A:  %8.2f ms\n", elapsed_time_ms);

  printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));

  if (d_data) hipFree(d_data);
  if (h_data) free(h_data);

  hipDeviceReset();

  return pass ? EXIT_SUCCESS : EXIT_FAILURE;
}
